#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_histogram.h"

#include <stdio.h>

__global__ void addCalculateKernel(const int *image, int* dest, unsigned int part_size)
{
	int tid = threadIdx.x;

	int image_start = part_size*tid;
	int image_end = image_start + part_size;

	int dest_start = ARR_SIZE*tid;
	int dest_end = dest_start + ARR_SIZE;

	// zero srtarting counters
	for (int j = dest_start; j < dest_end; j++) { dest[j] = 0; }

	for (int j = image_start; j < image_end; j++)
		dest[dest_start + image[j]]++;
}

__global__ void addMergeKernel(int* histogram, int* temp_arrays)
{
	int tid = threadIdx.x;
	// zero starting counters
	for (int i = 0; i < ARR_SIZE; i++) { histogram[i] = 0; }
	// merge results. each thread summarize one cell in each temp array
	for (int i = 0; i < NUM_OF_TH; i++)
		histogram[tid] += temp_arrays[ARR_SIZE*i + tid];
}

int* calculateHistogramm(int *image, unsigned int size)
{
	int *dev_image = 0;
	int *dev_dest_hist = 0;
	int *dev_histogram = 0;
	int* histogram = new int[ARR_SIZE]; ZeroGivenArray(histogram, ARR_SIZE);
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_image, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_histogram, ARR_SIZE * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_dest_hist, ARR_SIZE * sizeof(int)* NUM_OF_TH);
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_image, image, size * sizeof(int), hipMemcpyHostToDevice);
	// Launch a kernel on the GPU with one thread for each element.
	addCalculateKernel <<<1, NUM_OF_TH>>>(dev_image, dev_dest_hist, size/NUM_OF_TH);
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addCalculateKernel!\n", cudaStatus);
		goto Error;
	}

	addMergeKernel <<<1, ARR_SIZE>>>(dev_histogram, dev_dest_hist);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addCalculateKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(histogram, dev_histogram, ARR_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
Error:
	hipFree(dev_image);
	hipFree(dev_histogram);
	hipFree(dev_dest_hist);

	return histogram;
}